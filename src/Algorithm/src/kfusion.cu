#include "hip/hip_runtime.h"
#include "kfusion.h"
#include <hip/hip_vector_types.h>
#include "constant_parameters.h"
#include "utils.h"
#include "kernels.h"
#include "volume.h"
#include <thrust/device_vector.h>
#include<stdint.h>
#include<iostream>


//static bool firstAcquire = true;
dim3 imageBlock = dim3(32, 16);
dim3 raycastBlock = dim3(32, 8);

KFusion::KFusion(const kparams_t &par, sMatrix4 initPose)
    :params(par),
    _tracked(false)
{
    uint3 vr = make_uint3(params.volume_resolution.x,
                          params.volume_resolution.y,
                          params.volume_resolution.z);

    float3 vd = make_float3(params.volume_size.x,
                            params.volume_size.y,
                            params.volume_size.z);

    volume.init(vr,vd);
    newDataVol.init(vr,vd);

    pose = initPose;
    oldPose=pose;
    this->iterations.clear();
    for(auto it = params.pyramid.begin();it != params.pyramid.end(); it++)
    {    
        this->iterations.push_back(*it);
    }
    largestep=0.75*params.mu;
    inverseCam=getInverseCameraMatrix(params.camera);
    camMatrix=getCameraMatrix(params.camera);
    step = min(params.volume_size) / max(params.volume_resolution);
    viewPose = &pose;

    uint2 cs = make_uint2(params.computationSize.x, params.computationSize.y);
    std::cout<<"CS:"<<cs.x<<","<<cs.y<<std::endl;

    std::cout<<"KAM"<<std::endl;
    std::cout<<camMatrix<<std::endl;

    reduction.alloc(cs);
    vertex.alloc(cs);
    normal.alloc(cs);
    rawDepth.alloc(cs);
    depthImage.alloc(cs);
    rawRgb.alloc(cs);

    scaledDepth.resize(iterations.size());
    inputVertex.resize(iterations.size());
    inputNormal.resize(iterations.size());

    for (int i = 0; i < iterations.size(); ++i)
    {
        scaledDepth[i].alloc(cs >> i);
        inputVertex[i].alloc(cs >> i);
        inputNormal[i].alloc(cs >> i);
    }

    gaussian.alloc(make_uint2(radius * 2 + 1, 1));
    output.alloc(make_uint2(32, 8));
    //generate gaussian array
    generate_gaussian<<< 1,gaussian.size.x>>>(gaussian, delta, radius);
    dim3 grid = divup(dim3(volume.getResolution().x, volume.getResolution().y), imageBlock);

    printCUDAError();
    TICK("initVolume");
    initVolumeKernel<<<grid, imageBlock>>>(volume, make_float2(1.0f, 0.0f));
    TOCK();
    printCUDAError();

    //init new data volume
    initVolumeKernel<<<grid, imageBlock>>>(newDataVol, make_float2(1.0f, 0.0f));
    
    // render buffers
    renderModel.alloc(cs);
    //TODO better memory managment of covariance data
    covData.alloc(cs);
    
    if (printCUDAError())
    {
        hipDeviceReset();
        exit(1);
    }
}

KFusion::~KFusion()
{
    hipDeviceSynchronize();
    volume.release();
    
    reduction.release();
    normal.release();
    vertex.release();
    
    for(int i=0;i<inputVertex.size();i++)
    {
        inputVertex[i].release();
    }
    
    for(int i=0;i<inputNormal.size();i++)
    {
        inputNormal[i].release();
    }
     
    for(int i=0;i<scaledDepth.size();i++)
    {
        scaledDepth[i].release();
    }
    
    covData.release();
    rawDepth.release();
    rawRgb.release();
    depthImage.release();
    output.release();
    gaussian.release();
    
    renderModel.release();
    printCUDAError();
}

void KFusion::reset()
{
    dim3 grid = divup(dim3(volume.getResolution().x, volume.getResolution().y), imageBlock);
    initVolumeKernel<<<grid, imageBlock>>>(volume, make_float2(1.0f, 0.0f));
}

void KFusion::updateVolume()
{
    volume.updateData(newDataVol);
}


bool KFusion::preprocessing2(const float *inputDepth,const uchar3 *inputRgb)
{
    hipMemcpy(rawDepth.data(), inputDepth, params.inputSize.x * params.inputSize.y * sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(rawRgb.data(), inputRgb, params.inputSize.x * params.inputSize.y * sizeof(uchar3),hipMemcpyHostToDevice);

    dim3 grid = divup(make_uint2(params.inputSize.x, params.inputSize.y), imageBlock);
    TICK("bilateral_filter");
    bilateralFilterKernel<<<grid, imageBlock>>>(scaledDepth[0], rawDepth, gaussian, e_delta, radius);
    TOCK();

    return true;
}

bool KFusion::preprocessing(const ushort * inputDepth,const uchar3 *inputRgb)
{
    hipMemcpy(depthImage.data(), inputDepth, params.inputSize.x * params.inputSize.y * sizeof(ushort), hipMemcpyHostToDevice);
    TICK("mm2meters");
    mm2metersKernel<<<divup(rawDepth.size, imageBlock), imageBlock>>>(rawDepth, depthImage);
    TOCK();
    hipMemcpy(rawRgb.data(), inputRgb, params.inputSize.x * params.inputSize.y * sizeof(uchar3),hipMemcpyHostToDevice);

    // filter the input depth map
    dim3 grid = divup(make_uint2(params.computationSize.x, params.computationSize.y), imageBlock);
    TICK("bilateral_filter");
    bilateralFilterKernel<<<grid, imageBlock>>>(scaledDepth[0], rawDepth, gaussian, e_delta, radius);
    TOCK();        

    return true;
}

bool KFusion::tracking(uint frame)
{
    (void)frame;
    forcePose=false;
    std::vector<dim3> grids;
    for (int i = 0; i < iterations.size(); ++i)
        grids.push_back(divup(make_uint2(params.computationSize.x, params.computationSize.y) >> i, imageBlock));

    // half sample the input depth maps into the pyramid levels
    for (int i = 1; i < iterations.size(); ++i)
    {
        TICK("halfSampleRobust");
        halfSampleRobustImageKernel<<<grids[i], imageBlock>>>(scaledDepth[i], scaledDepth[i-1], e_delta * 3, 1);
        TOCK();
    }

    float4 k = make_float4(params.camera.x, params.camera.y, params.camera.z, params.camera.w);
    // prepare the 3D information from the input depth maps
    for (int i = 0; i < iterations.size(); ++i)
    {
        TICK("depth2vertex");
        depth2vertexKernel<<<grids[i], imageBlock>>>( inputVertex[i], scaledDepth[i], getInverseCameraMatrix(k / float(1 << i))); // inverse camera matrix depends on level
        TOCK();
        TICK("vertex2normal");
        vertex2normalKernel<<<grids[i], imageBlock>>>( inputNormal[i], inputVertex[i] );
        TOCK();
    }

    oldPose = pose;
    const sMatrix4 projectReference = camMatrix*inverse(sMatrix4(&raycastPose));

    for (int level = iterations.size() - 1; level >= 0; --level)
    {
        for (int i = 0; i < iterations[level]; ++i)
        {
            TICK("track");
            trackPose=pose;
            trackKernel<<<grids[level], imageBlock>>>( reduction,
                                                       inputVertex[level],
                                                       inputNormal[level],
                                                       vertex,
                                                       normal,
                                                       sMatrix4( & pose ),
                                                       projectReference,
                                                       dist_threshold,
                                                       normal_threshold);
            TOCK();
            TICK("reduce");
            reduceKernel<<<8, 112>>>( output.data(), reduction, inputVertex[level].size ); // compute the linear system to solve
            TOCK();
            hipDeviceSynchronize();// important due to async nature of kernel call

            TooN::Matrix<8, 32, float, TooN::Reference::RowMajor> values(output.data());
            for(int j = 1; j < 8; ++j)
                values[0] += values[j];

            if (updatePoseKernel(pose, output.data(), params.icp_threshold,this->deltaPose))
                break;
        }
    }

    return checkPoseKernel(pose, oldPose, output.data(), params.computationSize,track_threshold);
}

bool KFusion::raycasting(uint frame)
{
    if (frame > 2)
    {
        oldRaycastPose = raycastPose;
        raycastPose = pose;
        dim3 grid=divup(make_uint2(params.computationSize.x,params.computationSize.y),raycastBlock );
        TICK("raycast");
        raycastKernel<<<grid, raycastBlock>>>(vertex, normal, volume, sMatrix4(&raycastPose) * inverseCam,
                                              nearPlane,
                                              farPlane,
                                              step,
                                              largestep,frame);
        TOCK();
    }
    else
    {
        return false;
    }

    printCUDAError();

    return true;
}

void KFusion::integrateNewData(sMatrix4 p)
{
    dim3 grid=divup(dim3(newDataVol.getResolution().x, newDataVol.getResolution().y), imageBlock);
    //initVolumeKernel<<<grid, imageBlock>>>(newDataVol, make_float2(1.0f, 0.0f));


    integrateKernel<<<grid,imageBlock>>>(newDataVol,rawDepth,rawRgb,
                                         inverse(p),camMatrix,params.mu,maxweight );

}

void KFusion::integrateSlices(VolumeSlices &slices)
{

}



bool KFusion::integration(uint frame)
{
    //bool doIntegrate = checkPoseKernel(pose, oldPose, output.data(),params.computationSize, track_threshold);
    if (_tracked || frame <= 3)
    {
        printCUDAError();
        TICK("integrate");
        dim3 grid=divup(dim3(volume.getResolution().x, volume.getResolution().y), imageBlock);
        integrateKernel<<<grid, imageBlock>>>(volume,
                                              rawDepth,
                                              rawRgb,
                                              inverse(pose),
                                              camMatrix,
                                              params.mu,
                                              maxweight );

        TOCK();       
        return true;
    }

    return false;
}

bool KFusion::deIntegration(sMatrix4 p,const Host &depth,const Host &rgb)
{
    image_copy(rawDepth,depth, rawDepth.size.x*rawDepth.size.y*sizeof(float));
    image_copy(rawRgb,rgb, rawRgb.size.x*rawRgb.size.y*sizeof(uchar3));

    TICK("deintegrate");
    deIntegrateKernel<<<divup(dim3(volume.getResolution().x, volume.getResolution().y), imageBlock), imageBlock>>>(volume,
                                                                                           rawDepth,
                                                                                           rawRgb,
                                                                                           inverse(sMatrix4(&p)),
                                                                                           camMatrix,
                                                                                           params.mu,
                                                                                           maxweight);    
    TOCK();
    return true;
}

bool KFusion::reIntegration(sMatrix4 p,const Host &depth,const Host &rgb)
{    
    uint s = params.inputSize.x*params.inputSize.y;
    image_copy(rawDepth,depth, s*sizeof(float));
    image_copy(rawRgb,rgb, s*sizeof(uchar3));
    TICK("reintegrate");
    integrateKernel<<<divup(dim3(volume.getResolution().x, volume.getResolution().y), imageBlock), imageBlock>>>(volume,
                                                                                           rawDepth,
                                                                                           rawRgb,
                                                                                           inverse(sMatrix4(&p)),
                                                                                           camMatrix,
                                                                                           params.mu,
                                                                                           maxweight );
    TOCK();
    return true;
}

Image<float3, Host> KFusion::getAllVertex()
{
    Image<float3, Host> ret( make_uint2(params.inputSize.x, params.inputSize.y) );
    hipMemcpy(ret.data(), inputVertex[0].data(),
            params.inputSize.x * params.inputSize.y * sizeof(float3),
            hipMemcpyDeviceToHost);
    return ret;
}

Image<float3, Host> KFusion::getAllNormals()
{
    Image<float3, Host> ret( make_uint2(params.inputSize.x, params.inputSize.y) );
    hipMemcpy(ret.data(), inputNormal[0].data(),
            params.inputSize.x * params.inputSize.y * sizeof(float3),
            hipMemcpyDeviceToHost);
    return ret;
}

Image<TrackData, Host> KFusion::getTrackData()
{
    Image<TrackData, Host> trackData;
    trackData.alloc(reduction.size);

    hipMemcpy(trackData.data(), reduction.data(),reduction.size.x*reduction.size.y*sizeof(TrackData),hipMemcpyDeviceToHost);

    return trackData;
}


void KFusion::getVertices(std::vector<float3> &vertices)
{
    vertices.clear();
    short2 *hostData = (short2 *) malloc(volume.getResolution().x * volume.getResolution().y * volume.getResolution().z * sizeof(short2));

    if (hipMemcpy(hostData,
                   volume.getDataPtr(),
                   volume.getResolution().x *
                   volume.getResolution().y *
                   volume.getResolution().z *
                   sizeof(short2),
                   hipMemcpyDeviceToHost) != hipSuccess)
    {
        std::cerr << "Error reading volumetric representation data from the GPU. "<< std::endl;
        exit(1);
    }
    generateTriangles(vertices, volume, hostData);
    free(hostData);
}

void KFusion::renderVolume(uchar3 * out)
{
    dim3 grid=divup(renderModel.size,imageBlock);

    TICK("renderVolume");
    renderVolumeKernel2<<<grid,imageBlock>>>( renderModel,vertex,normal,light,ambient,nearPlane,farPlane);
    TOCK();
    

    hipMemcpy(out, renderModel.data(),
            params.computationSize.x * params.computationSize.y * sizeof(uchar3),
            hipMemcpyDeviceToHost);
}

Image<float, Host> KFusion::vertex2Depth()
{
    Image<float, Host> ret(params.inputSize);
//    Image<float, Device> model(params.inputSize);
    
//     dim3 grid=divup(model.size,imageBlock);
//    vertex2depthKernel<<<grid,imageBlock>>>( model,vertex,normal,nearPlane,farPlane,K);
    
//    hipMemcpy(ret.data(), model.data(),
//            params.inputSize.x * params.inputSize.y * sizeof(float),
//            hipMemcpyDeviceToHost);
    return ret;
}

float KFusion::compareRgb( )
{
    Image<float, Device> diff( make_uint2(params.inputSize.x, params.inputSize.y) );
    compareRgbKernel<<<divup(renderModel.size, imageBlock), imageBlock>>>( renderModel,rawRgb,diff);
    
    size_t size=params.inputSize.x*params.inputSize.y;
    thrust::device_ptr<float> diff_ptr(diff.data());
    thrust::device_vector<float> d_vec(diff_ptr,diff_ptr+size);
    float sum = thrust::reduce(d_vec.begin(), d_vec.end(), 0, thrust::plus<float>());

    float ret = sum/size;
    
    diff.release();
    return ret;
}

void KFusion::getImageProjection(sMatrix4 p, uchar3 *out)
{
    Image<float3, Device> vertexNew, normalNew;
    vertexNew.alloc(params.inputSize);
    normalNew.alloc(params.inputSize);

    dim3 grid=divup(params.inputSize,raycastBlock );
    //raycast from given pose
    printCUDAError();
    raycastKernel<<<grid, raycastBlock>>>(vertexNew, normalNew, volume, p * inverseCam,
                                         nearPlane,farPlane,step,largestep,1);
    
    hipDeviceSynchronize();
    printCUDAError();

    grid=divup(params.inputSize,imageBlock );
    renderRgbKernel<<<grid, imageBlock>>>( renderModel,volume,vertexNew,normalNew);

    hipMemcpy(out, renderModel.data(),
               params.inputSize.x * params.inputSize.y * sizeof(uchar3),
               hipMemcpyDeviceToHost);
    
    vertexNew.release();
    normalNew.release();
}

float KFusion::getWrongNormalsSize()
{

    dim3 grid=divup(make_uint2(params.computationSize.x,params.computationSize.y),raycastBlock );

    Image<int, Device> model;
    model.alloc(params.inputSize);

    wrongNormalsSizeKernel<<<grid, raycastBlock>>>( model,reduction );

    size_t size=params.inputSize.x*params.inputSize.y;

    thrust::device_ptr<int> diff_ptr(model.data());
    thrust::device_vector<int> d_vec(diff_ptr,diff_ptr+size);
    int sum = thrust::reduce(d_vec.begin(), d_vec.end(), 0, thrust::plus<int>());

    float ret = (float)sum/(params.computationSize.x*params.computationSize.y);
    
    model.release();
    return ret;
}

void KFusion::renderImage(uchar3 * out)
{
    TICK("renderVolume");
    hipDeviceSynchronize();
    dim3 grid=divup(renderModel.size, imageBlock);
    renderRgbKernel<<<grid, imageBlock>>>(renderModel,volume,vertex,normal);
    TOCK();

     hipMemcpy(out, renderModel.data(),
                params.computationSize.x * params.computationSize.y * sizeof(uchar3),
                hipMemcpyDeviceToHost);

}

void KFusion::renderTrack(uchar3 * out)
{
    dim3 grid=divup(renderModel.size, imageBlock);
    TICK("renderTrack");
    renderTrackKernel<<<grid, imageBlock>>>( renderModel, reduction );
    TOCK();
    hipMemcpy(out, renderModel.data(), params.inputSize.x * params.inputSize.y * sizeof(uchar3), hipMemcpyDeviceToHost);

    printCUDAError();
}

void KFusion::renderDepthFromVertex(uchar3 * out)
{
    Image<float, Device> depth;
    depth.alloc(rawDepth.size);
    dim3 grid=divup(renderModel.size, imageBlock);

    vertex2depthKernel<<<grid, imageBlock>>>( depth, inputVertex[0], camMatrix);
    renderDepthKernel<<<grid, imageBlock>>>( renderModel, depth, nearPlane, farPlane );
    hipMemcpy(out,renderModel.data(), params.inputSize.x * params.inputSize.y * sizeof(uchar3), hipMemcpyDeviceToHost);
}

void KFusion::renderDepth(uchar3 * out)
{
    TICK("renderDepthKernel");
    dim3 grid=divup(renderModel.size, imageBlock);
    renderDepthKernel<<<grid, imageBlock>>>( renderModel, rawDepth, nearPlane, farPlane );
    TOCK();
    hipMemcpy(out,renderModel.data(), params.inputSize.x * params.inputSize.y * sizeof(uchar3), hipMemcpyDeviceToHost);
}

bool KFusion::updatePoseKernel(sMatrix4 & pose, const float * output,float icp_threshold,sMatrix4 &deltaPose)
{
    // Update the pose regarding the tracking result
    TooN::Matrix<8, 32, const float, TooN::Reference::RowMajor> values(output);
    TooN::Vector<6> x = solve(values[0].slice<1, 27>());
    TooN::SE3<> delta(x);
    sMatrix4 deltaMat=tosMatrix4(delta);
    sMatrix4 delta4 = deltaMat * sMatrix4(&pose);

    pose.data[0].x = delta4.data[0].x;
    pose.data[0].y = delta4.data[0].y;
    pose.data[0].z = delta4.data[0].z;
    pose.data[0].w = delta4.data[0].w;
    pose.data[1].x = delta4.data[1].x;
    pose.data[1].y = delta4.data[1].y;
    pose.data[1].z = delta4.data[1].z;
    pose.data[1].w = delta4.data[1].w;
    pose.data[2].x = delta4.data[2].x;
    pose.data[2].y = delta4.data[2].y;
    pose.data[2].z = delta4.data[2].z;
    pose.data[2].w = delta4.data[2].w;
    pose.data[3].x = delta4.data[3].x;
    pose.data[3].y = delta4.data[3].y;
    pose.data[3].z = delta4.data[3].z;
    pose.data[3].w = delta4.data[3].w;

    // Return validity test result of the tracking
    if (norm(x) < icp_threshold)
    {
        deltaPose=deltaMat;
        return true;
    }
    return false;
}

bool KFusion::checkPoseKernel(sMatrix4 & pose,
                     sMatrix4 oldPose,
                     const float * output,
                     uint2 imageSize,
                     float track_threshold)
{
    if(forcePose)
    {
        _tracked=true;
        return true;
    }
    
    // Check the tracking result, and go back to the previous camera position if necessary
    // return true;
    TooN::Matrix<8, 32, const float, TooN::Reference::RowMajor> values(output);

    if ( (std::sqrt(values(0, 0) / values(0, 28)) > 2e-2) ||
         (values(0, 28) / (imageSize.x * imageSize.y) < track_threshold) )
    {
        pose = oldPose;
        _tracked=false;
        return false;
    }

    _tracked=true;
    poseInv=inverse(pose);
    return true;
}

void KFusion::getImageRaw(RgbHost &to) const
{

  uint s=(uint)rawRgb.size.x*rawRgb.size.y*sizeof(uchar3);
  to.alloc(rawRgb.size);
  hipMemcpy(to.data(), rawRgb.data(),s,hipMemcpyDeviceToHost);
//  to.size=rawRgb.size;
}

void KFusion::getDepthRaw(DepthHost &to) const
{
    uint s=(uint)rawDepth.size.x*rawDepth.size.y*sizeof(float);
    to.alloc(rawDepth.size);
    hipMemcpy(to.data(), rawDepth.data(),s,hipMemcpyDeviceToHost);
    to.size=rawDepth.size;
}

void KFusion::getIcpValues(Image<float3, Host> &depthVertex,
                             Image<float3, Host> &raycastVertex,
                             Image<float3, Host> &raycastNormals,
                             Image<TrackData, Host> &trackData) const
{
    uint s=(uint) (params.volume_size.x*params.volume_size.y);
    depthVertex.alloc(inputVertex[0].size);
    raycastVertex.alloc(vertex.size);
    raycastNormals.alloc(normal.size);
    trackData.alloc(reduction.size);
  
    hipMemcpy(depthVertex.data(), inputVertex[0].data(),s*sizeof(float3),hipMemcpyDeviceToHost);
    hipMemcpy(raycastVertex.data(), vertex.data(),s*sizeof(float3),hipMemcpyDeviceToHost);
    hipMemcpy(raycastNormals.data(), normal.data(),s*sizeof(float3),hipMemcpyDeviceToHost);
    hipMemcpy(trackData.data(), reduction.data(),reduction.size.x*reduction.size.y*sizeof(TrackData),hipMemcpyDeviceToHost);
}

float KFusion::getFitness()
{
    size_t size=reduction.size.x*reduction.size.y;
    thrust::device_ptr<TrackData> ptr=thrust::device_pointer_cast(reduction.data());
    TrackData d;
    d.result=1;
    int count = thrust::count(ptr,ptr+size,d);

    float ret=(float)count/size;
    return ret;
}


sMatrix6 KFusion::calculate_ICP_COV()
{
    sMatrix4 currPose=pose;
    sMatrix4 invPrevPose=inverse(oldPose);
    sMatrix4 delta=invPrevPose*currPose;

    sMatrix4 projectedReference = camMatrix*inverse(sMatrix4(&raycastPose));
    dim3 grid=divup(make_uint2(params.inputSize.x,params.inputSize.y),imageBlock );

    sMatrix6 initMat;
    for(int i=0;i<36;i++)
        initMat.data[i]=0.0;



    icpCovarianceFirstTerm<<<grid, imageBlock>>>(inputVertex[0],
                                                vertex,
                                                normal,
                                                reduction,
                                                covData,
                                                trackPose,
                                                projectedReference,
                                                delta,
                                                params.cov_big);
    
    hipDeviceSynchronize();    
    size_t size=covData.size.x*covData.size.y;
    thrust::device_ptr<sMatrix6> cov_ptr(covData.data());
    sMatrix6 d2J_dX2 = thrust::reduce(cov_ptr, cov_ptr+size, initMat, thrust::plus<sMatrix6>());

    icpCovarianceSecondTerm<<<grid, imageBlock>>>(inputVertex[0],
                                                  vertex,
                                                  normal,
                                                  reduction,
                                                  covData,
                                                  trackPose,
                                                  projectedReference,
                                                  delta,                                                  
                                                  params.cov_z,
                                                  params.cov_big);
    hipDeviceSynchronize();
    sMatrix6 covSecondTerm = thrust::reduce(cov_ptr, cov_ptr+size, initMat, thrust::plus<sMatrix6>());


    sMatrix6 d2J_dX2inv=inverse(d2J_dX2);
    sMatrix6 tmp=d2J_dX2inv * covSecondTerm;
    sMatrix6 icpCov= tmp * d2J_dX2inv;

    //make sure that covariance matrix is symetric.
    //small asymetries may occur due to numerical stability
    sMatrix6 ret;
    for(int i=0;i<6;i++)
    {
        for(int j=0;j<6;j++)
        {
            //eliminate NaN values
            if(icpCov(i,j)!=icpCov(i,j))
            {
                icpCov(i,j)=params.cov_big;
            }
            if(icpCov(j,i)!=icpCov(j,i))
            {
                icpCov(j,i)=params.cov_big;
            }
            float val=( icpCov(i,j) + icpCov(j,i))/2;
            ret(i,j)=val;
            ret(j,i)=val;

        }
    }

    return ret;
}



