#include "hip/hip_runtime.h"
#include"kernelscalls.h"
#include"kernels.h"
#include <thrust/device_vector.h>
#include "constant_parameters.h"
#include<iostream>


sMatrix6 calculatePoint2PointCov(const std::vector<float3> &vert,
                                 int vertSize,
                                 const std::vector<float3> &prevVert,
                                 int prevVertSize,
                                 //const int2 *corresp,
                                 const std::vector<int> &sourceCorr,
                                 const std::vector<int> &targetCorr,
                                 const sMatrix4 &tf,
                                 const kparams_t &params)
{
    float3 *vertGpu;
    hipMalloc(&vertGpu,vertSize*sizeof(float3));
    hipMemcpy(vertGpu,vert.data(),vertSize*sizeof(float3),hipMemcpyHostToDevice);

    float3 *prevVertGpu;
    hipMalloc(&prevVertGpu,prevVertSize*sizeof(float3));
    hipMemcpy(prevVertGpu,prevVert.data(),prevVertSize*sizeof(float3),hipMemcpyHostToDevice);

    size_t correspSize=sourceCorr.size();
    int *sourceCorrGpu;
    int err=hipMalloc(&sourceCorrGpu,correspSize*sizeof(int));
    hipMemcpy(sourceCorrGpu,sourceCorr.data(),correspSize*sizeof(int),hipMemcpyHostToDevice);

    int *targetCorrGpu;
    err=hipMalloc(&targetCorrGpu,correspSize*sizeof(int));
    hipMemcpy(targetCorrGpu,targetCorr.data(),correspSize*sizeof(int),hipMemcpyHostToDevice);
    
    sMatrix6 *covData;
    hipMalloc(&covData,correspSize*sizeof(sMatrix6));
    
    point2PointCovFirstTerm<<<(correspSize+256)/256, 256>>>(vertGpu,
                                                            vertSize,
                                                            prevVertGpu,
                                                            prevVertSize,
                                                            sourceCorrGpu,
                                                            targetCorrGpu,
                                                            correspSize,
                                                            tf,
                                                            covData,
                                                            params.cov_big);
    sMatrix6 initMat;
    for(int i=0;i<36;i++)
        initMat.data[i]=0.0;
    
    hipDeviceSynchronize();
    
    
    
    thrust::device_ptr<sMatrix6> cov_ptr(covData);
    sMatrix6 d2J_dX2 = thrust::reduce(cov_ptr, cov_ptr+correspSize, initMat, thrust::plus<sMatrix6>());

    //float cov_z=0.05;
    float cov_z=1;
    point2PointCovSecondTerm<<<(correspSize+256)/256, 256>>>(vertGpu,
                                                            vertSize,
                                                            prevVertGpu,
                                                            prevVertSize,
                                                            sourceCorrGpu,
                                                            targetCorrGpu,
                                                            correspSize,
                                                            tf,
                                                            cov_z,
                                                            covData,
                                                            params.cov_big);
    hipDeviceSynchronize();    
    sMatrix6 covSecondTerm = thrust::reduce(cov_ptr, cov_ptr+correspSize, initMat, thrust::plus<sMatrix6>());

//    std::cout<<"d2J_dX2"<<std::endl;
//    std::cout<<d2J_dX2<<std::endl;

    sMatrix6 d2J_dX2inv=inverse(d2J_dX2);


//    std::cout<<"d2J_dX2inv"<<std::endl;
//    std::cout<<d2J_dX2inv<<std::endl;

    sMatrix6 tmp=d2J_dX2inv * covSecondTerm;
    sMatrix6 icpCov= tmp * d2J_dX2inv;
    
    //make sure that covariance matrix is symetric.
    //small asymetries may occur due to numerical stability
    sMatrix6 ret;
    for(int i=0;i<6;i++)
    {
        for(int j=0;j<6;j++)
        {
            //eliminate NaN values
            if(icpCov(i,j)!=icpCov(i,j))
            {
                icpCov(i,j)=params.cov_big;
            }
            if(icpCov(j,i)!=icpCov(j,i))
            {
                icpCov(j,i)=params.cov_big;
            }
            float val=( icpCov(i,j) + icpCov(j,i))/2;
            ret(i,j)=val;
            ret(j,i)=val;

        }
    }
    
    hipFree(vertGpu);
    hipFree(prevVertGpu);
    hipFree(sourceCorrGpu);
    hipFree(targetCorrGpu);
    hipFree(covData);
    
    return ret;
}
